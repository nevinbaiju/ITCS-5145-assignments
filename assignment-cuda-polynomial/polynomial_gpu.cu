
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly, int degree,
			   int n, float * array) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n){
    
    float out = 0;
    float xtothepowerof = 1;
    float x = array[idx];
    for (int i=0; i<=degree; i++){
      out += xtothepowerof*poly[i];
      xtothepowerof *= x;
    }
    array[idx] = 2;
  }

}


int main (int argc, char* argv[]) {
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  float* d_array, * d_poly;
  hipMalloc(&d_array, n * sizeof(float));
  hipMalloc(&d_poly, (degree+1) * sizeof(float));

  hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_poly, poly, (degree+1) * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel to add vectors
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion<<<blocksPerGrid, threadsPerBlock>>>(d_poly, degree, n, d_array);
  
  hipMemcpy(array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  hipFree(d_array);
  hipFree(d_poly);

  return 0;
}
