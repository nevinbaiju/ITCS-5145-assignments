
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void addVectors(int* A, int* B, int* C, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        C[i] = A[i] + B[i];
    }
}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  int n = atoi(argv[1]); //TODO: atoi is an unsafe function

  float* h_in1 = new float[n];
  float* h_in2 = new float[n];
  float* h_out = new float[n];
  
  
  for (int i=0; i<n; ++i) {
    h_in1[i] = i;
    h_in2[i] = 100+i;
  }


  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  // Allocate memory on the device
  int* d_in1, * d_in2, * d_out;
  hipMalloc(&d_in1, n * sizeof(int));
  hipMalloc(&d_in2, n * sizeof(int));
  hipMalloc(&d_out, n * sizeof(int));

  // Copy vectors from host to device
  hipMemcpy(d_in1, h_in1, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_in2, h_in2, n * sizeof(int), hipMemcpyHostToDevice);

  // Launch kernel to add vectors
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_in1, d_in2, d_out, n);

  // Copy result vector from device to host
  hipMemcpy(h_out, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

  
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin);

  std::cout<<n<<" "<<totaltime.count()<<std::endl;

  delete[] h_in1;
  delete[] h_in2;
  delete[] h_out;

  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);

  return 0;
}
